#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include ""
#define SIZE 1024
#define THREADS_PER_BLOCK 16

__global__ void VectorAdd(int* a, int* b, int* c, int n)  //global ����� ����  device(gpu)�� �Լ� ����
{
	for (int i = 0; i < n; ++i)
		c[i] = a[i] + b[i];
}

int main()
{
	int* a, * b, * c;
	int* d_a, * d_b, * d_c;

	a = (int*)malloc(SIZE * sizeof(int));  // malloc�� ���� host(cpu)������ ������� Ȯ��
	b = (int*)malloc(SIZE * sizeof(int));
	c = (int*)malloc(SIZE * sizeof(int));

	hipMalloc(&d_a, (SIZE * sizeof(int))); //hipMalloc�� ���� device�� ��� ������ ������� Ȯ�� 
	hipMalloc(&d_b, (SIZE * sizeof(int)));
	hipMalloc(&d_c, (SIZE * sizeof(int)));

	for (int i = 0; i < SIZE; ++i) //host ���� �ʱⰪ ����
	{
		a[i] = i;
		b[i] = i;
		c[i] = 0;
	}

	hipMemcpy(d_a, a, (SIZE * sizeof(int)), hipMemcpyHostToDevice); // malloc���� ������ host���� ������� ũ��, device������ ����
	hipMemcpy(d_b, b, (SIZE * sizeof(int)), hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, (SIZE * sizeof(int)), hipMemcpyHostToDevice);


	VectorAdd << < SIZE / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> > (d_a, d_b, d_c, SIZE); //�����Լ� device����, <<< ����ϼ� , ��ϴ� ��뾲���� ��>>>

	hipMemcpy(a, d_a, (SIZE * sizeof(int)), hipMemcpyDeviceToHost);  //device���갪 host�� ����
	hipMemcpy(b, d_b, (SIZE * sizeof(int)), hipMemcpyDeviceToHost);
	hipMemcpy(c, d_c, (SIZE * sizeof(int)), hipMemcpyDeviceToHost);

	for (int i = 0; i < 10; ++i) //10ȸ ���� ����
		printf("c[%d] = %d\n", i, c[i]);

	free(a); //host ���� ������� ����
	free(b);
	free(c);

	hipFree(d_a); //device ���� ������� ����
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}