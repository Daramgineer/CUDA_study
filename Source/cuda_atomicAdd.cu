#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include ""
#include <stdio.h>
#include <stdlib.h>
#define _CRT_SECURE_NO_WARNINGS


#define SIZE 1024
#define THREADS_PER_BLOCK 16

__global__ void VectorDot(int* a, int* b, int* c)
{
	__shared__ int temp[THREADS_PER_BLOCK]; //block sheard mem�� temp�� ����
	int t_id = threadIdx.x + blockIdx.x * blockDim.x;
	temp[threadIdx.x] = a[t_id] * b[t_id]; //�� thread�� a, b ������ ����
	__syncthreads(); //block�� thread�� ���� ����ȭ

	int sum = 0;
	if (threadIdx.x == 0)
	{
		for (int i = 0; i < THREADS_PER_BLOCK; i++) //block�� thread �������� �ջ�
		{
			sum += temp[i];
		}
		atomicAdd(c, sum);
	}
}

int main()
{
	int* a, * b, * c;
	int* d_a, * d_b, * d_c;

	a = (int*)malloc(SIZE * sizeof(int));  // malloc�� ���� host(cpu)������ ������� Ȯ��
	b = (int*)malloc(SIZE * sizeof(int));
	c = (int*)malloc(sizeof(int));  // ������� �ϳ��� ���(a,b �������� �� ����).

	hipMalloc(&d_a, (SIZE * sizeof(int))); //hipMalloc�� ���� device�� ��� ������ ������� Ȯ�� 
	hipMalloc(&d_b, (SIZE * sizeof(int)));
	hipMalloc(&d_c, sizeof(int)); // ������� �ϳ��� ���(a,b �������� �� ����).

	for (int i = 0; i < SIZE; ++i) //host ���� �ʱⰪ ����
	{
		a[i] = i;
		b[i] = i;
	}

	hipMemcpy(d_a, a, (SIZE * sizeof(int)), hipMemcpyHostToDevice); // malloc���� ������ host���� ������� ũ��, device������ ����
	hipMemcpy(d_b, b, (SIZE * sizeof(int)), hipMemcpyHostToDevice);
	hipMemset(d_c, 0, sizeof(int));

	VectorDot << < 4, 16 >> > (d_a, d_b, d_c); //�����Լ� device����, <<< ����ϼ� , ��ϴ� ��뾲���� ��>>>
	hipDeviceSynchronize();

	hipMemcpy(c, d_c, sizeof(int), hipMemcpyDeviceToHost); //device������ host������ ����

	printf("Final Sum : %d\n", *c);

	free(a); //host ���� ������� ����
	free(b);
	free(c);

	hipFree(d_a); //device ���� ������� ����
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}