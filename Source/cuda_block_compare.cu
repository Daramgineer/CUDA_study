#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void loop()
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x; //thread num ����ȭ
	printf("This is iteration number %d\n", idx);
}

int main()
{
	clock_t st = clock(); //�����ð� üũ ����
	int N = 10;

	loop << <1, N >> > (); //single block (168ms)
	loop << <2, N / 2 >> > (); //multi block (152ms) 

	hipDeviceSynchronize(); //thread���� ����ȭ

	clock_t ed = clock(); //�����ð� üũ ����
	printf("time %u ms\n", ed - st); //�����ð� out
}